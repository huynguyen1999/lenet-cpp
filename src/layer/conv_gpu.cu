#include "hip/hip_runtime.h"
#include "conv_gpu.h"
#include <math.h>
#include <iostream>
#include <typeinfo>
#include <assert.h>

#define TILE_WIDTH 16

void ConvGpu::init()
{
    height_out = (1 + (height_in - height_kernel + 2 * pad_h) / stride);
    width_out = (1 + (width_in - width_kernel + 2 * pad_w) / stride);
    dim_out = height_out * width_out * channel_out;
    weight.resize(channel_in * height_kernel * width_kernel, channel_out);
    bias.resize(channel_out);
    grad_weight.resize(channel_in * height_kernel * width_kernel, channel_out);
    grad_bias.resize(channel_out);
    set_normal_random(weight.data(), weight.size(), 0, 0.01);
    set_normal_random(bias.data(), bias.size(), 0, 0.01);
}

__global__ void convolution_kernel(float *result, const float *input_data, const float *filter,
                                   const int num_samples, const int num_output_channels, const int num_input_channels,
                                   const int input_height, const int input_width, const int filter_size)
{
    const int output_height = input_height - filter_size + 1;
    const int output_width = input_width - filter_size + 1;

    int width_grid = ceil(1.0 * output_width / TILE_WIDTH);

    int batch_index = blockIdx.x;                                         // Batch number
    int output_feature_index = blockIdx.y;                                // Output feature index
    int row_index = (blockIdx.z / width_grid) * TILE_WIDTH + threadIdx.y; // Row index in the image matrix
    int col_index = (blockIdx.z % width_grid) * TILE_WIDTH + threadIdx.x; // Column index in the image matrix

    float result_accumulator = 0.0f;

    if (row_index < output_height && col_index < output_width)
    {
        for (int input_channel_index = 0; input_channel_index < num_input_channels; input_channel_index++) // Sum over all input channels
        {
            for (int filter_row = 0; filter_row < filter_size; filter_row++) // Filter of size filter_size x filter_size
            {
                for (int filter_col = 0; filter_col < filter_size; filter_col++)
                {
                    int input_row = row_index + filter_row;
                    int input_col = col_index + filter_col;
                    result_accumulator += input_data[(batch_index * (num_input_channels * input_height * input_width)) +
                                                     (input_channel_index * (input_height * input_width)) +
                                                     (input_row * input_width) +
                                                     input_col] *
                                          filter[(output_feature_index * (num_input_channels * filter_size * filter_size)) +
                                                 (input_channel_index * (filter_size * filter_size)) +
                                                 (filter_row * filter_size) +
                                                 filter_col];
                }
            }
        }
        result[(batch_index * (num_output_channels * output_height * output_width)) +
               (output_feature_index * (output_height * output_width)) +
               (row_index * output_width) +
               col_index] = result_accumulator;
    }
}

void ConvGpu::perform_convolution_gpu(float *output, const float *input, const float *filter,
                                      const int num_samples, const int num_output_channels, const int num_input_channels,
                                      const int input_height, const int input_width, const int filter_size)
{
    const int output_height = input_height - filter_size + 1;
    const int output_width = input_width - filter_size + 1;

    // Allocate device memory
    float *device_input, *device_output, *device_filter;
    CHECK(hipMalloc((void **)&device_input, num_samples * num_input_channels * input_height * input_width * sizeof(float)));         // Input feature map with num_input_channels
    CHECK(hipMalloc((void **)&device_output, num_samples * num_output_channels * output_height * output_width * sizeof(float)));     // Output feature map with num_output_channels
    CHECK(hipMalloc((void **)&device_filter, num_output_channels * num_input_channels * filter_size * filter_size * sizeof(float))); // Filter with size filter_size * filter_size

    // Copy input and filter data to device
    CHECK(hipMemcpy(device_input, input, num_samples * num_input_channels * input_height * input_width * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(device_filter, filter, num_output_channels * num_input_channels * filter_size * filter_size * sizeof(float), hipMemcpyHostToDevice));

    // Set the kernel dimensions and call the kernel
    int Z = ceil(1.0 * output_height / TILE_WIDTH) * ceil(1.0 * output_width / TILE_WIDTH);
    dim3 num_threads_per_block(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 num_blocks_in_grid(num_samples, num_output_channels, Z);

    // Launch the kernel
    GpuTimer timer;
    timer.Start();
    convolution_kernel<<<num_blocks_in_grid, num_threads_per_block>>>(device_output, device_input, device_filter, num_samples, num_output_channels, num_input_channels, input_height, input_width, filter_size);
    timer.Stop();
    std::cout << "\t- Layer has kernel time: " << timer.Elapsed() << " ms" << std::endl;

    // Copy the output back to the host
    CHECK(hipMemcpy(output, device_output, num_samples * num_output_channels * output_height * output_width * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK(hipFree(device_input));
    CHECK(hipFree(device_output));
    CHECK(hipFree(device_filter));
}

void ConvGpu::forward(const Matrix &bottom)
{
    GpuTimer timer;
    timer.Start();

    int n_sample = bottom.cols();
    top.resize(height_out * width_out * channel_out, n_sample);
    float *input_data = (float *)bottom.data();
    float *output_data = (float *)top.data();
    float *weight_data = (float *)weight.data();

    const int num_samples = n_sample;
    const int input_channel = channel_in;
    const int output_channel = channel_out;
    const int kernel_height = height_kernel; // Assuming width_kernel is also K

    if (input_channel == 1)
        std::cout << "Convolution c1 - GPU";
    else
        std::cout << "Convolution c3 - GPU";
    perform_convolution_gpu(output_data, input_data, weight_data,
                            num_samples, output_channel, input_channel,
                            height_in, width_in, kernel_height);

    // Stop layer timer
    timer.Stop();
    std::cout << "\t- Total layer time: " << timer.Elapsed() << " ms" << std::endl;
}

void ConvGpu::im2col(const Vector &image, Matrix &data_col)
{
    int hw_in = height_in * width_in;
    int hw_kernel = height_kernel * width_kernel;
    int hw_out = height_out * width_out;
    // im2col
    data_col.resize(hw_out, hw_kernel * channel_in);
    for (int c = 0; c < channel_in; c++)
    {
        Vector map = image.block(hw_in * c, 0, hw_in, 1); // c-th channel map
        for (int i = 0; i < hw_out; i++)
        {
            int step_h = i / width_out;
            int step_w = i % width_out;
            int start_idx = step_h * width_in * stride + step_w * stride; // left-top idx of window
            for (int j = 0; j < hw_kernel; j++)
            {
                int cur_col = start_idx % width_in + j % width_kernel - pad_w; // col after padding
                int cur_row = start_idx / width_in + j / width_kernel - pad_h;
                if (cur_col < 0 || cur_col >= width_in || cur_row < 0 ||
                    cur_row >= height_in)
                {
                    data_col(i, c * hw_kernel + j) = 0;
                }
                else
                {
                    int pick_idx = cur_row * width_in + cur_col;
                    data_col(i, c * hw_kernel + j) = map(pick_idx); // pick which pixel
                }
            }
        }
    }
}

void ConvGpu::col2im(const Matrix &data_col, Vector &image)
{
    int hw_in = height_in * width_in;
    int hw_kernel = height_kernel * width_kernel;
    int hw_out = height_out * width_out;
    // col2im
    image.resize(hw_in * channel_in);
    image.setZero();
    for (int c = 0; c < channel_in; c++)
    {
        for (int i = 0; i < hw_out; i++)
        {
            int step_h = i / width_out;
            int step_w = i % width_out;
            int start_idx = step_h * width_in * stride + step_w * stride; // left-top idx of window
            for (int j = 0; j < hw_kernel; j++)
            {
                int cur_col = start_idx % width_in + j % width_kernel - pad_w; // col after padding
                int cur_row = start_idx / width_in + j / width_kernel - pad_h;
                if (cur_col < 0 || cur_col >= width_in || cur_row < 0 ||
                    cur_row >= height_in)
                {
                    continue;
                }
                else
                {
                    // int pick_idx = start_idx + (j / width_kernel) * width_in + j % width_kernel;
                    int pick_idx = cur_row * width_in + cur_col;
                    image(c * hw_in + pick_idx) += data_col(i, c * hw_kernel + j); // pick which pixel
                }
            }
        }
    }
}

void ConvGpu::backward(const Matrix &bottom, const Matrix &grad_top)
{
    int n_sample = bottom.cols();
    grad_weight.setZero();
    grad_bias.setZero();
    grad_bottom.resize(height_in * width_in * channel_in, n_sample);
    grad_bottom.setZero();
    for (int i = 0; i < n_sample; i++)
    {

        // Forward no longer needs the next 3 rows
        Matrix data_col;
        im2col(bottom.col(i), data_col);
        data_cols[i] = data_col;

        // im2col of grad_top
        Matrix grad_top_i = grad_top.col(i);
        Matrix grad_top_i_col = Eigen::Map<Matrix>(grad_top_i.data(),
                                                   height_out * width_out, channel_out);
        // d(L)/d(w) = \sum{ d(L)/d(z_i) * d(z_i)/d(w) }
        grad_weight += data_cols[i].transpose() * grad_top_i_col;
        // d(L)/d(b) = \sum{ d(L)/d(z_i) * d(z_i)/d(b) }
        grad_bias += grad_top_i_col.colwise().sum().transpose();
        // d(L)/d(x) = \sum{ d(L)/d(z_i) * d(z_i)/d(x) } = d(L)/d(z)_col * w'
        Matrix grad_bottom_i_col = grad_top_i_col * weight.transpose();
        // col2im of grad_bottom
        Vector grad_bottom_i;
        col2im(grad_bottom_i_col, grad_bottom_i);
        grad_bottom.col(i) = grad_bottom_i;
    }
}

void ConvGpu::update(Optimizer &opt)
{
    Vector::AlignedMapType weight_vec(weight.data(), weight.size());
    Vector::AlignedMapType bias_vec(bias.data(), bias.size());
    Vector::ConstAlignedMapType grad_weight_vec(grad_weight.data(), grad_weight.size());
    Vector::ConstAlignedMapType grad_bias_vec(grad_bias.data(), grad_bias.size());

    opt.update(weight_vec, grad_weight_vec);
    opt.update(bias_vec, grad_bias_vec);
}

std::vector<float> ConvGpu::get_parameters() const
{
    std::vector<float> res(weight.size() + bias.size());
    // Copy the data of weights and bias to a long vector
    std::copy(weight.data(), weight.data() + weight.size(), res.begin());
    std::copy(bias.data(), bias.data() + bias.size(), res.begin() + weight.size());
    return res;
}

void ConvGpu::set_parameters(const std::vector<float> &param)
{
    if (static_cast<int>(param.size()) != weight.size() + bias.size())
        throw std::invalid_argument("Parameter size does not match");
    std::copy(param.begin(), param.begin() + weight.size(), weight.data());
    std::copy(param.begin() + weight.size(), param.end(), bias.data());
}

std::vector<float> ConvGpu::get_derivatives() const
{
    std::vector<float> res(grad_weight.size() + grad_bias.size());
    // Copy the data of weights and bias to a long vector
    std::copy(grad_weight.data(), grad_weight.data() + grad_weight.size(), res.begin());
    std::copy(grad_bias.data(), grad_bias.data() + grad_bias.size(),
              res.begin() + grad_weight.size());
    return res;
}