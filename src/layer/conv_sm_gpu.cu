#include "hip/hip_runtime.h"
#include "conv_sm_gpu.h"
#include <math.h>
#include <iostream>
#include <typeinfo>
#include <assert.h>

#define TILE_WIDTH_SHARED_C1 16
#define TILE_WIDTH_SHARED_C3 12

void ConvSmGpu::init()
{
    height_out = (1 + (height_in - height_kernel + 2 * pad_h) / stride);
    width_out = (1 + (width_in - width_kernel + 2 * pad_w) / stride);
    dim_out = height_out * width_out * channel_out;
    weight.resize(channel_in * height_kernel * width_kernel, channel_out);
    bias.resize(channel_out);
    grad_weight.resize(channel_in * height_kernel * width_kernel, channel_out);
    grad_bias.resize(channel_out);
    set_normal_random(weight.data(), weight.size(), 0, 0.01);
    set_normal_random(bias.data(), bias.size(), 0, 0.01);
}

__global__ void sm_convolution_kernel(float *output, const float *input, const float *kernel,
                                   const int num_samples, const int output_channel, const int input_channel,
                                   const int height, const int width, const int kernel_size)
{
    int TILE_WIDTH_SHARED;
    if (input_channel == 1)
    {
        TILE_WIDTH_SHARED = TILE_WIDTH_SHARED_C1;
    }
    else
    {
        TILE_WIDTH_SHARED = TILE_WIDTH_SHARED_C3;
    }

    extern __shared__ float shared_input[];

    const int H_out = height - kernel_size + 1;
    const int W_out = width - kernel_size + 1;

    int W_grid = ceil(1.0 * W_out / TILE_WIDTH_SHARED);

    int b = blockIdx.x;   // batch number
    int m = blockIdx.y;   // output feature
    int ty = threadIdx.y; // thread ID in the current TILE
    int tx = threadIdx.x;

    int h = (blockIdx.z / W_grid) * TILE_WIDTH_SHARED + ty; // row of the input image matrix
    int w = (blockIdx.z % W_grid) * TILE_WIDTH_SHARED + tx; // col of the input image matrix

    int startOfTile_h = (blockIdx.z / W_grid) * TILE_WIDTH_SHARED; // row of the input image matrix
    int startOfTile_w = (blockIdx.z % W_grid) * TILE_WIDTH_SHARED; // col of the input image matrix
    for (int c = 0; c < input_channel; c++)
    {
        for (int i = ty; i < TILE_WIDTH_SHARED + kernel_size - 1; i += TILE_WIDTH_SHARED)
        {
            for (int j = tx; j < TILE_WIDTH_SHARED + kernel_size - 1; j += TILE_WIDTH_SHARED)
            {
                if (startOfTile_h + i < height && startOfTile_w + j < width)
                {
                    shared_input[c * (TILE_WIDTH_SHARED + kernel_size - 1) * (TILE_WIDTH_SHARED + kernel_size - 1) + i * (TILE_WIDTH_SHARED + kernel_size - 1) + j] = input[b * (input_channel * height * width) + c * (height * width) + (startOfTile_h + i) * width + startOfTile_w + j];
                }
            }
        }
    }
    __syncthreads();

    if ((h < H_out) && (w < W_out))
    {
        float accum = 0.0f;
        for (int c = 0; c < input_channel; c++) // sum over all input features
        {
            for (int p = 0; p < kernel_size; p++) // KxK filter
                for (int q = 0; q < kernel_size; q++)
                    accum += shared_input[c * (TILE_WIDTH_SHARED + kernel_size - 1) * (TILE_WIDTH_SHARED + kernel_size - 1) + (p + ty) * (TILE_WIDTH_SHARED + kernel_size - 1) + (q + tx)] * kernel[m * (input_channel * kernel_size * kernel_size) + c * (kernel_size * kernel_size) + p * kernel_size + q];
        }
        output[b * (output_channel * H_out * W_out) + m * (H_out * W_out) + h * W_out + w] = accum;
    }
}

void ConvSmGpu::perform_convolution_gpu(float *output_data, const float *input_data, const float *weight_data,
                                                 const int num_samples, const int output_channel, const int input_channel,
                                                 const int height_in, const int width_in, const int kernel_height)
{
    int TILE_WIDTH_SHARED;
    if (input_channel == 1)
    {
        TILE_WIDTH_SHARED = TILE_WIDTH_SHARED_C1;
    }
    else
    {
        TILE_WIDTH_SHARED = TILE_WIDTH_SHARED_C3;
    }

    const int H_out = height_in - kernel_height + 1;
    const int W_out = width_in - kernel_height + 1;

    int inputSize = num_samples * input_channel * height_in * width_in * sizeof(float);
    int outputSize = num_samples * output_channel * H_out * W_out * sizeof(float);

    float *device_input, *device_output, *device_weight;

    CHECK(hipMalloc((void **)&device_input, inputSize));
    CHECK(hipMalloc((void **)&device_output, outputSize));
    CHECK(hipMalloc((void **)&device_weight, output_channel * input_channel * kernel_height * kernel_height * sizeof(float)));

    CHECK(hipMemcpy(device_input, input_data, inputSize, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(device_weight, weight_data, output_channel * input_channel * kernel_height * kernel_height * sizeof(float), hipMemcpyHostToDevice));

    dim3 numThreadsPerBlock, numBlocksInGrid;

    numThreadsPerBlock = dim3(TILE_WIDTH_SHARED, TILE_WIDTH_SHARED, 1);
    int shmem_size = input_channel * (TILE_WIDTH_SHARED + kernel_height - 1) * (TILE_WIDTH_SHARED + kernel_height - 1) * sizeof(float);
    numBlocksInGrid = dim3(num_samples, output_channel, ceil(1.0 * H_out / TILE_WIDTH_SHARED) * ceil(1.0 * W_out / TILE_WIDTH_SHARED));

    // Launch kernel
    GpuTimer timer;
    timer.Start();
    sm_convolution_kernel<<<numBlocksInGrid, numThreadsPerBlock, shmem_size>>>(device_output, device_input, device_weight, num_samples, output_channel, input_channel, height_in, width_in, kernel_height);
    timer.Stop();
    std::cout << "\tKernel Time: " << timer.Elapsed() << " ms" << std::endl;

    CHECK(hipMemcpy(output_data, device_output, outputSize, hipMemcpyDeviceToHost));

    CHECK(hipFree(device_input));
    CHECK(hipFree(device_output));
    CHECK(hipFree(device_weight));
}

void ConvSmGpu::forward(const Matrix &bottom)
{
    GpuTimer timer;
    timer.Start();

    int n_sample = bottom.cols();
    top.resize(height_out * width_out * channel_out, n_sample);
    float *input_data = (float *)bottom.data();
    float *output_data = (float *)top.data();
    float *weight_data = (float *)weight.data();

    const int num_samples = n_sample;
    const int input_channel = channel_in;
    const int output_channel = channel_out;
    const int kernel_height = height_kernel; // Assuming width_kernel is also K

    perform_convolution_gpu(output_data, input_data, weight_data,
                            num_samples, output_channel, input_channel,
                            height_in, width_in, kernel_height);

    // Stop layer timer
    timer.Stop();
    float duration_layer = timer.Elapsed();

    std::cout << "\t - Layer Time: " << duration_layer << " ms" << std::endl;
}

void ConvSmGpu::im2col(const Vector &image, Matrix &data_col)
{
    int hw_in = height_in * width_in;
    int hw_kernel = height_kernel * width_kernel;
    int hw_out = height_out * width_out;
    // im2col
    data_col.resize(hw_out, hw_kernel * channel_in);
    for (int c = 0; c < channel_in; c++)
    {
        Vector map = image.block(hw_in * c, 0, hw_in, 1); // c-th channel map
        for (int i = 0; i < hw_out; i++)
        {
            int step_h = i / width_out;
            int step_w = i % width_out;
            int start_idx = step_h * width_in * stride + step_w * stride; // left-top idx of window
            for (int j = 0; j < hw_kernel; j++)
            {
                int cur_col = start_idx % width_in + j % width_kernel - pad_w; // col after padding
                int cur_row = start_idx / width_in + j / width_kernel - pad_h;
                if (cur_col < 0 || cur_col >= width_in || cur_row < 0 ||
                    cur_row >= height_in)
                {
                    data_col(i, c * hw_kernel + j) = 0;
                }
                else
                {
                    int pick_idx = cur_row * width_in + cur_col;
                    data_col(i, c * hw_kernel + j) = map(pick_idx); // pick which pixel
                }
            }
        }
    }
}

void ConvSmGpu::col2im(const Matrix &data_col, Vector &image)
{
    int hw_in = height_in * width_in;
    int hw_kernel = height_kernel * width_kernel;
    int hw_out = height_out * width_out;
    // col2im
    image.resize(hw_in * channel_in);
    image.setZero();
    for (int c = 0; c < channel_in; c++)
    {
        for (int i = 0; i < hw_out; i++)
        {
            int step_h = i / width_out;
            int step_w = i % width_out;
            int start_idx = step_h * width_in * stride + step_w * stride; // left-top idx of window
            for (int j = 0; j < hw_kernel; j++)
            {
                int cur_col = start_idx % width_in + j % width_kernel - pad_w; // col after padding
                int cur_row = start_idx / width_in + j / width_kernel - pad_h;
                if (cur_col < 0 || cur_col >= width_in || cur_row < 0 ||
                    cur_row >= height_in)
                {
                    continue;
                }
                else
                {
                    // int pick_idx = start_idx + (j / width_kernel) * width_in + j % width_kernel;
                    int pick_idx = cur_row * width_in + cur_col;
                    image(c * hw_in + pick_idx) += data_col(i, c * hw_kernel + j); // pick which pixel
                }
            }
        }
    }
}

void ConvSmGpu::backward(const Matrix &bottom, const Matrix &grad_top)
{
    int n_sample = bottom.cols();
    grad_weight.setZero();
    grad_bias.setZero();
    grad_bottom.resize(height_in * width_in * channel_in, n_sample);
    grad_bottom.setZero();
    for (int i = 0; i < n_sample; i++)
    {

        // Forward no longer needs the next 3 rows
        Matrix data_col;
        im2col(bottom.col(i), data_col);
        data_cols[i] = data_col;

        // im2col of grad_top
        Matrix grad_top_i = grad_top.col(i);
        Matrix grad_top_i_col = Eigen::Map<Matrix>(grad_top_i.data(),
                                                   height_out * width_out, channel_out);
        // d(L)/d(w) = \sum{ d(L)/d(z_i) * d(z_i)/d(w) }
        grad_weight += data_cols[i].transpose() * grad_top_i_col;
        // d(L)/d(b) = \sum{ d(L)/d(z_i) * d(z_i)/d(b) }
        grad_bias += grad_top_i_col.colwise().sum().transpose();
        // d(L)/d(x) = \sum{ d(L)/d(z_i) * d(z_i)/d(x) } = d(L)/d(z)_col * w'
        Matrix grad_bottom_i_col = grad_top_i_col * weight.transpose();
        // col2im of grad_bottom
        Vector grad_bottom_i;
        col2im(grad_bottom_i_col, grad_bottom_i);
        grad_bottom.col(i) = grad_bottom_i;
    }
}

void ConvSmGpu::update(Optimizer &opt)
{
    Vector::AlignedMapType weight_vec(weight.data(), weight.size());
    Vector::AlignedMapType bias_vec(bias.data(), bias.size());
    Vector::ConstAlignedMapType grad_weight_vec(grad_weight.data(), grad_weight.size());
    Vector::ConstAlignedMapType grad_bias_vec(grad_bias.data(), grad_bias.size());

    opt.update(weight_vec, grad_weight_vec);
    opt.update(bias_vec, grad_bias_vec);
}

std::vector<float> ConvSmGpu::get_parameters() const
{
    std::vector<float> res(weight.size() + bias.size());
    // Copy the data of weights and bias to a long vector
    std::copy(weight.data(), weight.data() + weight.size(), res.begin());
    std::copy(bias.data(), bias.data() + bias.size(), res.begin() + weight.size());
    return res;
}

void ConvSmGpu::set_parameters(const std::vector<float> &param)
{
    if (static_cast<int>(param.size()) != weight.size() + bias.size())
        throw std::invalid_argument("Parameter size does not match");
    std::copy(param.begin(), param.begin() + weight.size(), weight.data());
    std::copy(param.begin() + weight.size(), param.end(), bias.data());
}

std::vector<float> ConvSmGpu::get_derivatives() const
{
    std::vector<float> res(grad_weight.size() + grad_bias.size());
    // Copy the data of weights and bias to a long vector
    std::copy(grad_weight.data(), grad_weight.data() + grad_weight.size(), res.begin());
    std::copy(grad_bias.data(), grad_bias.data() + grad_bias.size(),
              res.begin() + grad_weight.size());
    return res;
}